#include "hip/hip_runtime.h"
/* Copyright (C) 2015 INRA
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be included
 * in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
 * OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <mitm/mitm.hpp>
#include "internal.hpp"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>

namespace mitm {

typedef signed char int8;

__global__
void
initialize(int8 *x, std::size_t N)
{
  x[threadIdx.x] = threadIdx.x;
}


void
gpgpu_properties_show()
{
  int nDevices;

  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
    std::cerr << "GPGPU initialization fail: "
              << hipGetErrorString(err) << "\n";
  } else {
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);

      std::cout << "Device " << i
                << "\n---------------------------------\n"
                << "\nMajor revision number:         " << prop.major
                << "\nMinor revision number:         " << prop.minor
                << "\nName:                          " << prop.name
                << "\nTotal global memory:           " << prop.totalGlobalMem
                << " (" << (prop.totalGlobalMem / (1024.0 * 1024)) << " Gb)"
                << "\nTotal shared memory per block: "
                << prop.sharedMemPerBlock
                << "\nTotal registers per block:     " << prop.regsPerBlock
                << "\nWarp size:                     " << prop.warpSize
                << "\nMaximum memory pitch:          " << prop.memPitch
                << " (" << (prop.memPitch / (1024.0 * 1024)) << " Gb)"
                << "\nMaximum threads per block:     "
                << prop.maxThreadsPerBlock
                << "\nMaximum dimension 0 of block:  "
                << prop.maxThreadsDim[0]
                << "\nMaximum dimension 1 of block:  "
                << prop.maxThreadsDim[1]
                << "\nMaximum dimension 2 of block:  "
                << prop.maxThreadsDim[2]
                << "\nMaximum dimension 0 of grid:   " << prop.maxGridSize[0]
                << "\nMaximum dimension 1 of grid:   " << prop.maxGridSize[1]
                << "\nMaximum dimension 2 of grid:   " << prop.maxGridSize[2]
                << "\nClock rate:                    " << prop.clockRate
                << "\nTotal constant memory:         " << prop.totalConstMem
                << "\nTexture alignment:             "
                << prop.textureAlignment
                << "\nConcurrent copy and execution: "
                << (prop.deviceOverlap ? "Yes" : "No")
                << "\nNumber of multiprocessors:     "
                << prop.multiProcessorCount
                << "\nKernel execution timeout:      "
                << (prop.kernelExecTimeoutEnabled ? "Yes" : "No")
                << '\n';
    }
  }
}

result
heuristic_algorithm_gpgu(const SimpleState &s, index limit,
                         mitm::real kappa, mitm::real delta,
			 mitm::real theta)
{
(void)s;
(void)limit;
(void)kappa;
(void)delta;
  std::vector <int8> s_x(100, 0);

  std::cout << "Run in GPGPU\n";
  gpgpu_properties_show();

  int8 *x;

  hipMalloc((void**)&x, s_x.size() * sizeof(int8_t));
  hipMemcpy(x, s_x.data(), s_x.size() * sizeof(int8_t), hipMemcpyHostToDevice);

  int block_size = 4;
  int n_blocks = s_x.size() /block_size + (s_x.size() % block_size == 0 ? 0:1);

  // initialize<<<n_blocks, block_size>>>(x, s_x.size());

  hipMemcpy(s_x.data(), x, s_x.size() * sizeof(int8_t), hipMemcpyDeviceToHost);
  hipFree(x);

  return result();
}

}
